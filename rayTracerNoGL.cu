#include "hip/hip_runtime.h"

#include <iostream>
#include "Reader.h"
#include "cutil_math.h"
#include <>
#include <hip/device_functions.h>


//forward declarations
uint hash(uint seed);

//These are the device sphere and triangle pointers. 
Sphere *dev_sphere_ptr;
Triangle *dev_tri_ptr;
//These two variables are the device pointers to min and max of AABB
float3 *dev_AABB_ptr;

//These numbers come directly from smallPT
//had to scale everything down by a factor of 10 to reduce artifacts.
//all spheres go in this list, here. This is messy. 
//spheres and triangles will eventually be moved to the .cpp file, and used through
//pointers in the .cu file
Sphere spheres[] = {
	{ 1e4f, { 1e4f + .10f, 4.08f, 8.16f }, { 0.0f, 0.0f, 0.0f }, { 0.25f, 0.75f, 0.25f }, DIFF }, //Left 
	{ 1e4f, { -1e4f + 9.90f, 4.08f, 8.16f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f }, DIFF }, //Right 
	{ 1e4f, { 5.00f, 4.08f, 1e4f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Back 
	{ 1e3f, { 5.00f, 4.08f, -1e4f + 60.00f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, DIFF }, //Front 
	{ 1e4f, { 5.00f, 1e4f, 8.16f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Bottom 
	{ 1e4f, { 5.00f, -1e4f + 8.16f, 8.16f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Top 
	{ 0.5f, { 2.0f, 0.5f, 4.70f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, SPEC }, // small sphere 1
	{ 1.65f, { 7.30f, 1.65f, 7.80f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, REFR }, // small sphere 2
	{ 60.0f, { 5.00f, 68.16f - .05f, 8.16f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f }, DIFF }  // Light
};

//LOADING DATA TO DEVICE DRAM////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////
//From here down, there are a series of methods that are called to load various things to 
//Device DRAM. This includes spheres, triangles, meshes, and AABB's. It also includes code
//That allows for storing the data in a different kind of device RAM (ie. texture memory, which
//is cached.
//////////////////////////////////////////////////////////////////////////////////////////

//this function loads the spheres defined above into DRAM
void loadSpheresToMemory(Sphere *sph_list, int numberofspheres){
	size_t numspheres = numberofspheres * sizeof(Sphere);
	printf("\mLoading %d bytes for %d spheres,\n", numspheres, numberofspheres);
	hipMalloc((void **)&dev_sphere_ptr, numspheres);//void** cast is so hipMalloc will accept the address of sphere pointer as parameter
	hipMemcpy(dev_sphere_ptr, &sph_list[0], numspheres, hipMemcpyHostToDevice);
}

//this function loads the triangles defined above into DRAM
void loadTrisToMemory(Triangle *tri_list, int numberoftris){
	size_t numtris = numberoftris * sizeof(Triangle);
	printf("Loading %d bytes for %d triangles,\n", numtris, numberoftris);
	hipMalloc((void **)&dev_tri_ptr, numtris); //void** cast is so hipMalloc will accept the address of triangle pointer as parameter
	hipMemcpy(dev_tri_ptr, &tri_list[0], numtris, hipMemcpyHostToDevice);
}

//this function loads an entire mesh's worth of triangles to dev_tri_ptr
void loadMeshToMemory(loadingTriangle *tri_list, int numberoftris){
	
	//I really hope this doesn't crash.This was done to avoid having the new Triangle[numberoftris] call because we didn't have a global 
	//constructor for Triangle.
	//Allocate memory for bytes that hold Triangle and that will be copied to dev_tri_ptr
	void* triangles = malloc((sizeof(Triangle)*numberoftris));
	//copy from tri_list to triangles
	memcpy(triangles, tri_list, sizeof(Triangle)*numberoftris);
	//our trianglelist is a pointer to the first element in triangles (i.e. our triangles!)
	Triangle* trianglelist = (Triangle*)triangles; 
	//This copies element by element from loadingTriangle into Triangle. It adds col, emit, and refl_t as hard-coded value. Eventally, this will be read
	//from .matl files. 
	for (int i = 0; i < numberoftris; i++){
		trianglelist[i].v1 = make_float3(tri_list[i].v1.x, tri_list[i].v1.y, tri_list[i].v1.z);
		trianglelist[i].v2 = make_float3(tri_list[i].v2.x, tri_list[i].v2.y, tri_list[i].v2.z);
		trianglelist[i].v3 = make_float3(tri_list[i].v3.x, tri_list[i].v3.y, tri_list[i].v3.z);
		trianglelist[i].col = make_float3(0.6,0.9,0.6);
		trianglelist[i].emit = make_float3(0, 0, 0);
		trianglelist[i].refl = DIFF;
	}
	printf("Loading mesh made of %d triangles for %d bytes\n\n", numberoftris, numberoftris*sizeof(Triangle));
	
	//Note - This will over-write the other triangles stored at &dev_tri_ptr
	size_t numtris = numberoftris * sizeof(Triangle);
	hipMalloc((void**)&dev_tri_ptr, numtris);
	hipMemcpy(dev_tri_ptr, &trianglelist[0], numtris, hipMemcpyHostToDevice);
	delete[] trianglelist;
}

//this function loads the AABB to dev_min_ptr and dev_max_ptr
//with the bytes of data at &min and &max. 
//This cude is CLUSTERFUCKed. Casts on casts on casts 
void loadAABBtoMemory(float3 *AABB){
	size_t box_bytes = 2 * sizeof(float3);
	hipMalloc((void**)&dev_AABB_ptr, box_bytes);
	hipMemcpy(dev_AABB_ptr, &AABB[0], box_bytes, hipMemcpyHostToDevice);
	printf("Successfully loaded AABB with:\nmin: (%.2f, %.2f, %.2f)\nmax: (%.2f, %.2f, %.2f)\n", AABB[0].x, AABB[0].y, AABB[0].z, AABB[1].x, AABB[1].y, AABB[1].z);
}

__device__ inline bool intersectBoundingBox(const Ray &r, float3* AABB){
	float3 invdir = make_float3(1.f / r.dir.x, 1.f / r.dir.y, 1.f / r.dir.z);

	float t1 = (AABB[0].x - r.origin.x) * invdir.x;
	float t2 = (AABB[1].x - r.origin.x) * invdir.x;
	float t3 = (AABB[0].y - r.origin.y) * invdir.y;
	float t4 = (AABB[1].y - r.origin.y) * invdir.y;
	float t5 = (AABB[0].z - r.origin.z) * invdir.z;
	float t6 = (AABB[1].z - r.origin.z) * invdir.z;

	float tmin = fmaxf(fmaxf(fminf(t1, t2),fminf(t3, t4)), fminf(t5, t6));
	float tmax = fminf(fminf(fmaxf(t1, t2), fmaxf(t3, t4)), fmaxf(t5, t6));
	//printf("min = (%.2f, %.2f, %.2f), max = (%.2f, %.2f, %.2f)\ntmin = %.2f, tmax=%.2f\n", AABB[0].x, AABB[0].y, AABB[0].z, AABB[1].x, AABB[1].y, AABB[1].z, tmin, tmax);
	//if tmax < 0, ray intersects AABB but in the inverse direction (i.e. it's behind us)
	if (tmax < 0)
	{
		//t = tmax;
		return false;
	}

	//if tmin  > tmax, ray doesn't intersect AABB
	if (tmin > tmax)
	{
		//t = tmax;
		return false;
	}
	//t = tmin;
	return true;
}
//This function is an inline implementation that intersects a list of triangles - tri_list . This intersect method goes through the device constant memory where
//the mesh is stored. It returns true if the ray intersects this entire mesh at all. 
__device__ inline void intersectListOfTriangles(const Ray &r, float &t, int &id, Triangle* tri_list, int numtris, int numspheres){
	float tprime = 1e15;
	for (int i = 0; i < numtris; i++){
		if ((tprime = tri_list[i].intersectTri(r)) && tprime < t){
			t = tprime;
			id = i + numspheres;
		}
	}
}

//World description: 9 spheres that form a modified Cornell box. this can be kept in const GPU memory (for now)
__device__ inline bool intersectScene(const Ray &r, float &t, int &id, Sphere *sphere_list, int numspheres, Triangle *tri_list, int numtris, float3 *AABB){
	float tprime;
	float inf = 1e15f;
	t = inf; //initialize t to infinite distance
	for (int i = 0; i < numspheres; i++){//cycle through all spheres, until i<0
		if ((tprime = sphere_list[i].intersectSphere(r)) && tprime < t){//new intersection is closer than previous closest
			t = tprime;
			id = i; //store hit sphere by ID (array index)
		}
	}
	//0 through 8 for ID represent spheres 1 through 9
	//the next ID's correspond to triangles
	//before testing all the triangles in the mesh, first test intersection with the bounding box defined by min and max (AABB[0], AABB[1]
	
	bool use_AABB = true;
	////this section of code calls inline functions that do the intersecting. This should makei  easier to add other *intersection modules* including using texture memory and 
	if (use_AABB){
		if (intersectBoundingBox(r, AABB)){
			intersectListOfTriangles(r, t, id, tri_list, numtris, numspheres);
		}
	}
	
	else{
		intersectListOfTriangles(r, t, id, tri_list, numtris, numspheres);
	}

	//if hit occured, t is > 0 and < inf.
	return t < inf;
}


//This function calculates radiance at a given ray, returned by a color
//This solves the rendering equation : outgoing radiance (pixel, point, w/e) = emitted radiance + reflected radiance
//reflected radiance is integral of incoming radiance from hemisphere above point about surface normal, multiplied
//by reflectence function of material, weighted by cosine incidence angle (Lambert's cosine law)
//Inputs: ray to calculate radiance along, and seeds for random num generation.
//Output: color at point.
__device__ float3 radiance(Ray &r, hiprandState *randstate, Sphere *sphere_list, Triangle *tri_list, int numtris, float3 *AABB){
	float3 accumulated = make_float3(0.f, 0.f, 0.f); //accumulated ray color for each iteration of loop
	float3 mask = make_float3(1.f, 1.f, 1.f);
	int numspheres = 9;

	Refl_t refltype;

	float3 hitpt;
	float3 norm;
	float3 hitnorm;
	float3 hitobj_color;
	float3 hitobj_emit;

	//ray bounce loop, will use Russian Roulette later
	for (int bounces = 0; bounces < 10; bounces++){ //this iterative loop replaces recursive path tracing method; max depth is 4 bounces (no RR)
		float t; //distance to hitt
		int id = 0; //index of hit 
		float3 d; //next ray direction

		if (!intersectScene(r, t, id, sphere_list, numspheres, tri_list, numtris, AABB))
			return make_float3(0.f, 0.f, 0.f); //return background color of 0 if no hit

		//if the loop gets to here, we have hit. compute hit point and surface normal
		if (id < numspheres){
			//identify which sphere was hit, calculate normal and transfer material properties
			//const Sphere &hitobj = sphere_list[id];
			hitpt = r.origin + r.dir*t;
			hitnorm = normalize(hitpt - sphere_list[id].cent); //surface normal
			//reverse normal if going through object - used to determine where we are for refraction
			float ntest = dot(hitnorm, r.dir);
			norm = (ntest < 0 ? hitnorm : hitnorm * -1);
			
			//material info
			hitobj_color = sphere_list[id].col;
			hitobj_emit = sphere_list[id].emit;

			accumulated += mask*hitobj_emit; //add emitted light to accumulated color, masked by previous bounces
			refltype = sphere_list[id].refl;
		}
		else{ //hit item was not a sphere, therefore it was a triangle.
			hitpt = r.origin + r.dir*t;
			hitnorm = tri_list[id - numspheres].get_Normal(hitpt);
			float ntest = dot(hitnorm, r.dir);
			norm = (ntest < 0 ? hitnorm : hitnorm * -1);

			//material info
			hitobj_color = tri_list[id - numspheres].col;
			hitobj_emit = tri_list[id - numspheres].emit;

			accumulated += mask*hitobj_emit; //add emitted light to accumulated color, masked by previous bounces
			refltype = tri_list[id - numspheres].refl;
		}

		//here we branch based on Refl_t; for now all are diffuse. Get a new random ray in hemisphere above hitnorm
		if (refltype == DIFF){
			//first create 2 random numbers
			float r1 = 2 * M_PI*hiprand_uniform(randstate); //random number on unit circle
			float r2 = hiprand_uniform(randstate); //random number for elevation 
			float r2sq = sqrtf(r2);

			//must get local orthonormal coordinates u v and w at hitpt for new random ray dir
			float3 w = norm;
			//based on abs.val of w's x component (> or < .1) cross w with straight along y (0,1,0) or along x(1,0,0)
			float3 u = normalize(cross((fabs(w.x)>0.1f ? make_float3(0.f, 1.f, 0.f) : make_float3(1.f, 0.f, 0.f)), w));
			float3 v = cross(w, u);
			//now compute random ray direction on this hemisphere, in polar coordinates
			//note, cosine weighted importance sampling favors ray directions closer to the surf normal
			d = normalize(u * cosf(r1) * r2sq + v * sinf(r1) * r2sq + w * sqrtf(1.f - r2));
			//new ray origin is at hitpt, shifted a small amount along normal to prevent self-intersection


			mask *= hitobj_color; //multiply mask by object color for next bounce
			//apply Lambert's cosine law to get weighted importance sampling 
			mask *= dot(d, norm);
			hitpt += norm * .001f;
			mask *= 2.f; //divide by material pdf
		}
		else if (refltype == SPEC){//compute reflected ray direction
			d = reflect(r.dir, hitnorm);
			hitpt += norm * .001f;
			mask *= hitobj_color;
			//pdf = 1, don't need to divide by PDF
		}
		//REFR reflective type represents glass: index of refraction 1.4
		//Consider creating an index system for materials
		else {

			bool into = dot(norm, hitnorm) > 0; // is ray entering or leaving refractive material?
			float nc = 1.0f;  // Index of Refraction air
			float nt = 1.5f;  // Index of Refraction glass/water
			float nnt = into ? nc / nt : nt / nc;  // IOR ratio of refractive materials
			float ddn = dot(r.dir, norm);
			float cos2t = 1.0f - nnt*nnt * (1.f - ddn*ddn);

			if (cos2t < 0.0f) // total internal reflection 
			{
				d = reflect(r.dir, hitnorm); //d = r.dir - 2.0f * n * dot(n, r.dir);
			}
			else // cos2t > 0
			{
				// compute direction of transmission ray
				float3 tdir = normalize(r.dir * nnt - hitnorm * ((into ? 1 : -1) * (ddn*nnt + sqrtf(cos2t))));

				float R0 = (nt - nc)*(nt - nc) / (nt + nc)*(nt + nc);
				float c = 1.f - (into ? -ddn : dot(tdir, hitnorm));
				float Re = R0 + (1.f - R0) * c * c * c * c * c; //shlick's approx of Fresnel equation, probability of reflection
				float Tr = 1 - Re;
				float P = .25f + .5f * Re;
				float RP = Re / P;
				float TP = Tr / (1.f - P);

				// randomly choose reflection or transmission ray
				if (hiprand_uniform(randstate) < Re) // reflection ray
				{
					mask *= RP;
					d = reflect(r.dir, hitnorm);
				}
				else // transmission ray
				{
					mask *= TP;
					d = tdir;
				}
			}
		}
		//russian roulette
		if (bounces > 3){
			float P = getMax(mask);
			if (hiprand_uniform(randstate) > P)
				break;
			mask *= 1 / P;
		}
		r.origin = hitpt;
		r.dir = d;
	}

	return accumulated;
}

//hashing function to get seed for curandDevice
//this fast hash method was developed by Thomas Wang
//this is used to re-seed hiprand every sample
uint hash(uint seed){
	seed = (seed ^ 61) ^ (seed >> 16);
	seed *= 9;
	seed = seed ^ (seed >> 4);
	seed *= 0x27d4eb2d;
	seed = seed ^ (seed >> 15);
	return seed;
}
//this is the main rendering kernel that can be called from the CPU, runs in parallel on CUDA threads.
//each pixel runs in parallel
__global__ void render_kernel(float3 *out, uint hashedSampleNumber, Sphere *sphere_list, Triangle *tri_list, int numtris, float3 *AABB){
	//assign thread to every pixel
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	//printf("Pixeli, j = %d, %d\n", pixel_i, pixel_j);
	unsigned int i = (YRES - y - 1)*XRES + x; //get current pixel index from thread index
	
	hiprandState randstate;
	//int globalThreadID = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(x+y, 0, 0, &randstate);

	float3 cam_origin = make_float3(5.f, 5.2f, 19.56f);
	float3 cam_up = normalize(make_float3(0.f, 1, 0));
	float3 cam_target = make_float3(5.f, 5.2f, 0.f);
	Camera rayCaster = Camera(cam_origin, cam_target, cam_up);
	float3 col = make_float3(0.f, 0.f, 0.f); //reset for each pixel

	for (int s = 0; s < SAMPLES; s++){
		//primary ray dir, randomly jittered by a small amount (will be changed when there's a better camera struct)
		col = col + radiance(rayCaster.computeCameraRay(x, y, &randstate), &randstate, sphere_list, tri_list, numtris, AABB);// (1.f / SAMPLES);
	}
	//write rgb value of pixel to image buffer on GPU, clamped on [0.0f, 1.0f]
	float cor = (1.f / SAMPLES); //cor = correction: we want the average color
	out[i] = make_float3(clamp(col.x*cor, 0.f, 1.f), clamp(col.y*cor, 0.f, 1.f), clamp(col.z*cor, 0.f, 1.f));
}

//this wrapper function is used when the cpp main file calls the render kernel
void renderKernelWrapper(float3* out_host, int numspheres, loadingTriangle* tri_list, int numtris, float3* AABB){
	float3* out_dvc;

	hipMalloc(&out_dvc, XRES * YRES * sizeof(float3));

	loadSpheresToMemory(spheres, numspheres);
	loadMeshToMemory(tri_list, numtris);
	loadAABBtoMemory(AABB);
	
	dim3 block(16, 16, 1);
	dim3 grid(XRES / block.x, YRES / block.y, 1);

	printf("\nLaunchng render_kernel for %d samples\n", SAMPLES);
	render_kernel << <grid, block >> > (out_dvc, hash(124), dev_sphere_ptr, dev_tri_ptr, numtris, dev_AABB_ptr);

	hipMemcpy(out_host, out_dvc, XRES * YRES * sizeof(float3), hipMemcpyDeviceToHost);
	hipFree(out_dvc);

	hipDeviceSynchronize();

}